#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2024, NVIDIA Corporation. All rights reserved.
//
// This work is made available under the Nvidia Source Code License-NC.
// To view a copy of this license, visit
// https://nvlabs.github.io/gbrl/license.html
//
//////////////////////////////////////////////////////////////////////////////
#include <hip/hip_runtime.h>
#include <>

#include "cuda_utils.h"
#include "cuda_types.h"

hipError_t allocateCudaMemory(void** device_ptr, size_t size, const std::string& error_message) {
    hipError_t error = hipMalloc(device_ptr, size);
    if (error != hipSuccess) {
        size_t free_mem, total_mem;
        hipMemGetInfo(&free_mem, &total_mem);

        std::cerr << "CUDA Allocation Error: " << error_message << " - " 
                  << hipGetErrorString(error) << " when trying to allocate "
                  << (size / (1024.0 * 1024.0)) << " MB." << std::endl;

        std::cerr << "Free memory: " << (free_mem / (1024.0 * 1024.0)) << " MB, "
                  << "Total memory: " << (total_mem / (1024.0 * 1024.0)) << " MB." << std::endl;
    }
    return error;
}

void get_grid_dimensions(int n_elements, int& blocks, int& threads_per_block) {
    const int max_threads_per_block = THREADS_PER_BLOCK; // You can adjust this based on your GPU

    if (n_elements < WARP_SIZE) {
        threads_per_block = WARP_SIZE;
        blocks = 1;
        return;
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    blocks = n_elements / max_threads_per_block + 1;

    threads_per_block = 1 << static_cast<int>(ceilf(log2f(static_cast<float>(n_elements) / blocks)));
    threads_per_block = (threads_per_block > deviceProp.maxThreadsPerBlock) ? deviceProp.maxThreadsPerBlock : threads_per_block;
}